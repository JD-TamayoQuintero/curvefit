//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: main.cu
//
// GPU Coder version                    : 1.4
// CUDA/C/C++ source code generated on  : 06-Dec-2020 18:02:22
//

//***********************************************************************
// This automatically generated example CUDA main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************

// Include Files
#include "main.h"
#include "MEDT.h"
#include "MEDT_emxAPI.h"
#include "MEDT_terminate.h"

// Function Declarations
static boolean_T argInit_boolean_T();
static emxArray_boolean_T *c_argInit_UnboundedxUnbounded_b();
static void main_MEDT();

// Function Definitions

//
// Arguments    : void
// Return Type  : boolean_T
//
static boolean_T argInit_boolean_T()
{
  return false;
}

//
// Arguments    : void
// Return Type  : emxArray_boolean_T *
//
static emxArray_boolean_T *c_argInit_UnboundedxUnbounded_b()
{
  emxArray_boolean_T *result;
  int loopUpperBound;
  int idx0;
  int b_loopUpperBound;
  int idx1;

  // Set the size of the array.
  // Change this size to the value that the application requires.
  result = emxCreate_boolean_T(2, 2);

  // Loop over the array to initialize each element.
  loopUpperBound = result->size[0U];
  for (idx0 = 0; idx0 < loopUpperBound; idx0++) {
    b_loopUpperBound = result->size[1U];
    for (idx1 = 0; idx1 < b_loopUpperBound; idx1++) {
      // Set the value of the array element.
      // Change this value to the value that the application requires.
      result->data[idx0 + result->size[0] * idx1] = argInit_boolean_T();
    }
  }

  return result;
}

//
// Arguments    : void
// Return Type  : void
//
static void main_MEDT()
{
  emxArray_real_T *newimage;
  emxArray_boolean_T *image;
  emxInitArray_real_T(&newimage, 2);

  // Initialize function 'MEDT' input arguments.
  // Initialize function input argument 'image'.
  image = c_argInit_UnboundedxUnbounded_b();

  // Call the entry-point 'MEDT'.
  MEDT(image, newimage);
  emxDestroyArray_real_T(newimage);
  emxDestroyArray_boolean_T(image);
}

//
// Arguments    : int argc
//                const char * const argv[]
// Return Type  : int
//
int main(int, const char * const [])
{
  // The initialize function is being called automatically from your entry-point function. So, a call to initialize is not included here. 
  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_MEDT();

  // Terminate the application.
  // You do not need to do this more than one time.
  MEDT_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
