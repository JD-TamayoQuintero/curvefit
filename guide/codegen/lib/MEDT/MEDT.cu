#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: MEDT.cu
//
// GPU Coder version                    : 1.4
// CUDA/C/C++ source code generated on  : 06-Dec-2020 18:02:22
//

// Include Files
#include "MEDT.h"
#include "MEDT_emxutil.h"
#include "MWCudaDimUtility.h"
#include "MWLaunchParametersUtilities.h"
#include <cstring>

// Function Declarations
static __global__ void MEDT_kernel1(const emxArray_boolean_T *image, int i,
  emxArray_real_T *newimage);
static void gpuEmxFree_boolean_T(emxArray_boolean_T *inter);
static void gpuEmxFree_real_T(emxArray_real_T *inter);
static void gpuEmxMemcpyCpuToGpu_boolean_T(const emxArray_boolean_T *cpu,
  emxArray_boolean_T *inter, emxArray_boolean_T *gpu);
static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu);
static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter);
static void gpuEmxReset_boolean_T(emxArray_boolean_T *inter);
static void gpuEmxReset_real_T(emxArray_real_T *inter);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const emxArray_boolean_T *image
//                int i
//                emxArray_real_T *newimage
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void MEDT_kernel1(const
  emxArray_boolean_T *image, int i, emxArray_real_T *newimage)
{
  unsigned int threadId;
  unsigned int threadStride;
  unsigned int idx;
  int vlen;
  boolean_T maxval;
  int b_i;
  int c_i;
  int j;
  long long loopEnd;
  unsigned int tmpIndex;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  threadStride = static_cast<unsigned int>(mwGetTotalThreadsLaunched());
  loopEnd = (static_cast<long long>((image->size[1] - 1)) + 1LL) * (static_cast<
    long long>((i - 1)) + 1LL) - 1LL;
  for (idx = threadId; idx <= static_cast<unsigned int>(loopEnd); idx +=
       threadStride) {
    j = static_cast<int>((idx % static_cast<unsigned int>(image->size[1])));
    tmpIndex = (idx - static_cast<unsigned int>(j)) / static_cast<unsigned int>
      (image->size[1]);
    b_i = static_cast<int>(tmpIndex);
    vlen = image->size[1];
    maxval = image->data[b_i];
    for (c_i = 0; c_i <= vlen - 2; c_i++) {
      if (static_cast<int>(maxval) < static_cast<int>(image->data[b_i +
           image->size[0] * (c_i + 1)])) {
        maxval = image->data[b_i + image->size[0] * (c_i + 1)];
      }
    }

    if (static_cast<int>(image->data[b_i + image->size[0] * j]) == static_cast<
        int>(maxval)) {
      newimage->data[b_i + newimage->size[0] * j] = 255.0;
    } else {
      newimage->data[b_i + newimage->size[0] * j] = 0.0;
    }
  }
}

//
// Arguments    : emxArray_boolean_T *inter
// Return Type  : void
//
static void gpuEmxFree_boolean_T(emxArray_boolean_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

//
// Arguments    : emxArray_real_T *inter
// Return Type  : void
//
static void gpuEmxFree_real_T(emxArray_real_T *inter)
{
  hipFree(inter->data);
  hipFree(inter->size);
}

//
// Arguments    : const emxArray_boolean_T *cpu
//                emxArray_boolean_T *inter
//                emxArray_boolean_T *gpu
// Return Type  : void
//
static void gpuEmxMemcpyCpuToGpu_boolean_T(const emxArray_boolean_T *cpu,
  emxArray_boolean_T *inter, emxArray_boolean_T *gpu)
{
  int actualSize;
  int i;
  int allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(boolean_T));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(boolean_T),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

//
// Arguments    : const emxArray_real_T *cpu
//                emxArray_real_T *inter
//                emxArray_real_T *gpu
// Return Type  : void
//
static void gpuEmxMemcpyCpuToGpu_real_T(const emxArray_real_T *cpu,
  emxArray_real_T *inter, emxArray_real_T *gpu)
{
  int actualSize;
  int i;
  int allocatingSize;
  if (inter->numDimensions < cpu->numDimensions) {
    inter->numDimensions = cpu->numDimensions;
    hipFree(inter->size);
    hipMalloc(&inter->size, inter->numDimensions * sizeof(int));
  } else {
    inter->numDimensions = cpu->numDimensions;
  }

  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  if (inter->allocatedSize < actualSize) {
    if (inter->canFreeData) {
      hipFree(inter->data);
    }

    allocatingSize = cpu->allocatedSize;
    if (allocatingSize < actualSize) {
      allocatingSize = actualSize;
    }

    inter->allocatedSize = allocatingSize;
    inter->canFreeData = true;
    hipMalloc(&inter->data, inter->allocatedSize * sizeof(double));
  }

  hipMemcpy(inter->data, cpu->data, actualSize * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(inter->size, cpu->size, cpu->numDimensions * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu, inter, 32ULL, hipMemcpyHostToDevice);
}

//
// Arguments    : emxArray_real_T *cpu
//                emxArray_real_T *inter
// Return Type  : void
//
static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T
  *inter)
{
  int actualSize;
  int i;
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }

  hipMemcpy(cpu->data, inter->data, actualSize * sizeof(double),
             hipMemcpyDeviceToHost);
  hipMemcpy(cpu->size, inter->size, inter->numDimensions * sizeof(int),
             hipMemcpyDeviceToHost);
}

//
// Arguments    : emxArray_boolean_T *inter
// Return Type  : void
//
static void gpuEmxReset_boolean_T(emxArray_boolean_T *inter)
{
  std::memset(inter, 0, sizeof(emxArray_boolean_T));
}

//
// Arguments    : emxArray_real_T *inter
// Return Type  : void
//
static void gpuEmxReset_real_T(emxArray_real_T *inter)
{
  std::memset(inter, 0, sizeof(emxArray_real_T));
}

//
// Arguments    : const emxArray_boolean_T *image
//                emxArray_real_T *newimage
// Return Type  : void
//
void MEDT(const emxArray_boolean_T *image, emxArray_real_T *newimage)
{
  int i;
  int i1;
  emxArray_boolean_T *gpu_image;
  dim3 grid;
  dim3 block;
  boolean_T validLaunchParams;
  emxArray_real_T *gpu_newimage;
  boolean_T newimage_dirtyOnGpu;
  emxArray_boolean_T inter_image;
  emxArray_real_T inter_newimage;
  hipMalloc(&gpu_newimage, 32ULL);
  gpuEmxReset_real_T(&inter_newimage);
  hipMalloc(&gpu_image, 32ULL);
  gpuEmxReset_boolean_T(&inter_image);
  newimage_dirtyOnGpu = false;

  //  numero de datos
  i = image->size[0];
  i1 = newimage->size[0] * newimage->size[1];
  newimage->size[0] = image->size[0];
  newimage->size[1] = image->size[1];
  emxEnsureCapacity_real_T(newimage, i1);
  validLaunchParams = mwGetLaunchParameters(static_cast<double>((((image->size[1]
    - 1) + 1LL) * ((i - 1) + 1LL))), &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    gpuEmxMemcpyCpuToGpu_boolean_T(image, &inter_image, gpu_image);
    gpuEmxMemcpyCpuToGpu_real_T(newimage, &inter_newimage, gpu_newimage);
    MEDT_kernel1<<<grid, block>>>(gpu_image, i, gpu_newimage);
    newimage_dirtyOnGpu = true;
  }

  if (newimage_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(newimage, &inter_newimage);
  }

  gpuEmxFree_boolean_T(&inter_image);
  hipFree(gpu_image);
  gpuEmxFree_real_T(&inter_newimage);
  hipFree(gpu_newimage);
}

//
// File trailer for MEDT.cu
//
// [EOF]
//
